#include "hip/hip_runtime.h"
#include "pointcloud.h"
#include "../tools.cuh"


__global__ void kernelCounting(Chunk *grid, Vector3 *cloud, PointCloudMetadata metadata, uint16_t gridSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= metadata.pointAmount) {
        return;
    }
    Vector3 point = cloud[index];

    auto gridIndex = tools::calculateGridIndex(point, metadata, gridSize);

    atomicAdd(&(grid + gridIndex)->count, 1);
}

void PointCloud::initialPointCounting(uint32_t initialDepth) {

    itsInitialDepth = initialDepth;
    itsGridSize = pow(2, initialDepth);
    auto cellAmount = static_cast<uint32_t>(pow(itsGridSize, 3));

    // Create the counting grid
    itsGrid.push_back(make_unique<CudaArray<Chunk>>(cellAmount));
    hipMemset (itsGrid[0]->devicePointer(), 0, cellAmount * sizeof(uint32_t));

    dim3 grid, block;
    tools::create1DKernel(block, grid, itsData->pointCount());

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernelCounting <<<  grid, block >>> (
            itsGrid[0]->devicePointer(),
                    itsData->devicePointer(),
                    itsMetadata,
                    itsGridSize);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    spdlog::info("'initialPointCounting' took {:f} [ms]", milliseconds);
}