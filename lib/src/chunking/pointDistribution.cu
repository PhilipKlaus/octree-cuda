#include "hip/hip_runtime.h"
#include "pointcloud.h"
#include "../tools.cuh"
#include "../timing.cuh"


__global__ void kernelDistributing(
        Chunk *grid,
        Vector3 *cloud,
        uint64_t *treeData,
        PointCloudMetadata metadata,
        uint64_t gridSize
        ) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= metadata.pointAmount) {
        return;
    }
    Vector3 point = cloud[index];

    auto gridIndex = tools::calculateGridIndex(point, metadata, gridSize);

    uint64_t dst = gridIndex;
    bool isFinished = grid[dst].isFinished;

    while(!isFinished) {
        dst = grid[dst].parentChunkIndex;
        isFinished = grid[dst].isFinished;
    }

    uint64_t i = atomicAdd(&(grid[dst].indexCount), 1);
    treeData[grid[dst].treeIndex + i] = index;
}

void PointCloud::distributePoints() {

    dim3 grid, block;
    tools::create1DKernel(block, grid, itsData->pointCount());

    tools::KernelTimer timer;
    timer.start();
    kernelDistributing <<<  grid, block >>> (
            itsGrid->devicePointer(),
            itsData->devicePointer(),
            itsTreeData->devicePointer(),
            itsMetadata,
            itsGridBaseSideLength);
    timer.stop();

    spdlog::info("'distributePoints' took {:f} [ms]", timer.getMilliseconds());
}
