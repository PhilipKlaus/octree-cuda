#include "hip/hip_runtime.h"
#include "pointcloud.h"
#include "../tools.cuh"
#include "../timing.cuh"


__global__ void kernelDistributing(
        Chunk *grid,
        Vector3 *cloud,
        Vector3 *treeData,
        uint64_t *tmpIndexRegister,
        PointCloudMetadata metadata,
        uint64_t gridSize
        ) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= metadata.pointAmount) {
        return;
    }
    Vector3 point = cloud[index];

    auto gridIndex = tools::calculateGridIndex(point, metadata, gridSize);

    uint64_t dst = gridIndex;
    bool isFinished = grid[dst].isFinished;

    while(!isFinished) {
        dst = grid[dst].parentChunkIndex;
        isFinished = grid[dst].isFinished;
    }

    uint64_t i = atomicAdd(&tmpIndexRegister[dst], 1);
    treeData[grid[dst].chunkDataIndex + i] = cloud[index];
}

void PointCloud::distributePoints() {

    // Create temporary indexRegister for assigning an index for each point within its chunk area
    auto tmpIndexRegister = make_unique<CudaArray<uint64_t>>(itsCellAmount, "tmpIndexRegister");

    // Calculate kernel dimensions
    dim3 grid, block;
    tools::create1DKernel(block, grid, itsCloudData->pointCount());

    // Call distribution kernel
    tools::KernelTimer timer;
    timer.start();
    kernelDistributing <<<  grid, block >>> (
            itsGrid->devicePointer(),
            itsCloudData->devicePointer(),
            itsChunkData->devicePointer(),
            tmpIndexRegister->devicePointer(),
            itsMetadata,
            itsGridBaseSideLength);
    timer.stop();

    // Manually delete the original point cloud data on GPU -> it is not needed anymore
    itsCloudData.reset();

    spdlog::info("'distributePoints' took {:f} [ms]", timer.getMilliseconds());
}
