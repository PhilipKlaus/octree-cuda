#include "hip/hip_runtime.h"
#include "pointcloud.h"
#include "tools.cuh"

__global__ void kernelCounting(Chunk *grid, Vector3 *cloud, uint32_t pointCount, Vector3 posOffset, Vector3 size, Vector3 minimum, uint16_t gridSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= pointCount) {
        return;
    }
    Vector3 point = cloud[index];

    // Copied from OctreeConverter
    float dGridSize = gridSize;
    auto X = int32_t((point.x - posOffset.x) / 1);
    auto Y = int32_t((point.y - posOffset.y) / 1);
    auto Z = int32_t((point.z - posOffset.z) / 1);

    float ux = (float(X) * 1 + posOffset.x - minimum.x) / size.x;
    float uy = (float(Y) * 1 + posOffset.y - minimum.y) / size.y;
    float uz = (float(Z) * 1 + posOffset.z - minimum.z) / size.z;

    uint64_t ix = int64_t( fmin (dGridSize * ux, dGridSize - 1.0f));
    uint64_t iy = int64_t( fmin (dGridSize * uy, dGridSize - 1.0f));
    uint64_t iz = int64_t( fmin (dGridSize * uz, dGridSize - 1.0f));

    uint64_t gridIndex = ix + iy * gridSize + iz * gridSize * gridSize;

    atomicAdd(&(grid + gridIndex)->count, 1);
}

void PointCloud::initialPointCounting(uint32_t initialDepth, PointCloudMetadata metadata) {
    itsMetadata = metadata;

    itsInitialDepth = initialDepth;
    itsGridSize = pow(2, initialDepth);
    auto cellAmount = static_cast<uint32_t>(pow(itsGridSize, 3));

    // Create the counting grid
    itsGrid.push_back(make_unique<CudaArray<Chunk>>(cellAmount));
    hipMemset (itsGrid[0]->devicePointer(), 0, cellAmount * sizeof(uint32_t));

    dim3 grid, block;
    create1DKernel(block, grid, itsData->pointCount());

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernelCounting <<<  grid, block >>> (
            itsGrid[0]->devicePointer(),
                    itsData->devicePointer(),
                    itsData->pointCount(),
                    metadata.cloudOffset,
                    metadata.boundingBox.size(),
                    metadata.boundingBox.minimum,
                    itsGridSize);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    spdlog::info("'initialPointCounting' took {:f} [ms]", milliseconds);
}