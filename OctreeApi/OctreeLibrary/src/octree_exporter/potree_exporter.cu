#include "hip/hip_runtime.h"
#include "potree_exporter.cuh"
#include <iomanip>
#include <iostream>
#include <json.hpp>
#include <list>
#include <unordered_map>

constexpr char METADATA_FILE_NAME[]    = "/metadata.json";
constexpr char HIERARCHY_FILE_NAME[]   = "/hierarchy.bin";
constexpr char POINT_FILE_NAME[]       = "/octree.bin";
constexpr char POTREE_DATA_VERSION[]   = "2.0";
constexpr char POTREE_DATA_ENCODING[]  = "default";
constexpr uint8_t HIERARCHY_NODE_BYTES = 22;
constexpr uint8_t HIERARCHY_STEP_SIZE  = 100;
constexpr uint8_t HIERARCHY_DEPTH      = 20;

// POSITIONS
constexpr char POSITION_NAME[]          = "position";
constexpr char POSITION_TYPE[]          = "int32";
constexpr uint8_t POSITION_ELEMENTS     = 3;
constexpr uint8_t POSITION_ELEMENT_SIZE = sizeof (uint32_t);
constexpr uint8_t POSITION_SIZE         = POSITION_ELEMENT_SIZE * 3;

// COLORS
constexpr char COLOR_NAME[]          = "rgb";
constexpr char COLOR_TYPE[]          = "uint16";
constexpr uint8_t COLOR_ELEMENTS     = 3;
constexpr uint8_t COLOR_ELEMENT_SIZE = sizeof (uint16_t);
constexpr uint8_t COLOR_SIZE         = COLOR_ELEMENT_SIZE * 3;


PotreeExporter::PotreeExporter (
        const PointCloud& pointCloud,
        const std::shared_ptr<Chunk[]>& octree,
        OctreeMetadata metadata,
        PointCloudMetadata cloudMetadata,
        SubsampleMetadata subsamplingMetadata) :
        OctreeExporter (pointCloud, octree, metadata, cloudMetadata, subsamplingMetadata)
{}

void PotreeExporter::exportOctree (const std::string& path)
{
    this->itsPointsExported = 0;
    itsExportFolder         = path;
    itsExportedNodes        = 0;
    createBinaryHierarchyFiles ();
    createMetadataFile ();
}

void PotreeExporter::createBinaryHierarchyFiles ()
{
    std::ofstream pointFile;
    pointFile.open (itsExportFolder + POINT_FILE_NAME, std::ios::binary);
    std::ofstream hierarchyFile;
    hierarchyFile.open (itsExportFolder + HIERARCHY_FILE_NAME, std::ios::binary);

    breathFirstExport (pointFile, hierarchyFile);
    pointFile.write (reinterpret_cast<const char*> (this->itsOutputBuffer.get ()), this->itsOutputBufferSize);

    pointFile.close ();
    hierarchyFile.close ();
}

void PotreeExporter::breathFirstExport (std::ofstream& pointFile, std::ofstream& hierarchyFile)
{
    std::unordered_map<uint32_t, bool> discoveredNodes;
    std::list<uint32_t> toVisit;

    discoveredNodes[this->getRootIndex ()] = true;
    toVisit.push_back (this->getRootIndex ());

    while (!toVisit.empty ())
    {
        auto node = toVisit.front ();
        toVisit.pop_front ();

        uint8_t bitmask     = getChildMask (node);
        uint8_t type        = bitmask == 0 ? 1 : 0;
        uint64_t byteOffset = this->getDataIndex (node) * (3 * (sizeof (uint32_t) + sizeof (uint16_t)));
        uint64_t byteSize   = this->getPointsInNode (node) * (3 * (sizeof (uint32_t) + sizeof (uint16_t)));
        HierarchyFileEntry entry{type, bitmask, this->getPointsInNode (node), byteOffset, byteSize};
        hierarchyFile.write (reinterpret_cast<const char*> (&entry), sizeof (HierarchyFileEntry));

        this->itsPointsExported += this->getPointsInNode (node);
        ++itsExportedNodes;

        for (auto i = 0; i < 8; ++i)
        {
            int childNode = this->getChildNodeIndex (node, i);
            if (childNode != -1 && discoveredNodes.find (childNode) == discoveredNodes.end () &&
                this->isFinishedNode (childNode))
            {
                discoveredNodes[childNode] = true;
                toVisit.push_back (childNode);
            }
        }
    }
}


inline uint8_t PotreeExporter::getChildMask (uint32_t nodeIndex)
{
    uint8_t bitmask = 0;
    for (auto i = 0; i < 8; i++)
    {
        int childNodeIndex = this->getChildNodeIndex (nodeIndex, i);
        if (childNodeIndex != -1 && this->isFinishedNode (childNodeIndex))
        {
            bitmask = bitmask | (1 << i);
        }
    }
    return bitmask;
}

void PotreeExporter::createMetadataFile ()
{
    // Prepare metadata for export
    auto bbCubic = this->itsCloudMetadata.bbCubic;
    auto scale   = this->itsCloudMetadata.scale;
    auto spacing = (bbCubic.max.x - bbCubic.min.x) / this->itsSubsampleMetadata.subsamplingGrid;

    // Common metadata
    nlohmann::ordered_json metadata;
    metadata["version"]     = POTREE_DATA_VERSION;
    metadata["name"]        = "GpuPotreeConverter";
    metadata["description"] = "AIT Austrian Institute of Technology";
    metadata["points"]      = this->itsPointsExported;
    metadata["projection"]  = "";
    metadata["flags"][0]    = this->itsSubsampleMetadata.useReplacementScheme ? "REPLACING" : "ADDITIVE";
    if (this->itsSubsampleMetadata.performAveraging)
    {
        metadata["flags"][1] = "AVERAGING";
    }
    metadata["hierarchy"]["firstChunkSize"] = itsExportedNodes * HIERARCHY_NODE_BYTES;
    metadata["hierarchy"]["stepSize"]       = HIERARCHY_STEP_SIZE;
    metadata["hierarchy"]["depth"]          = HIERARCHY_DEPTH;
    metadata["offset"]                      = {0, 0, 0}; // We are not shifting the cloud
    metadata["scale"]                       = {scale.x, scale.y, scale.z};
    metadata["spacing"]                     = spacing;
    metadata["boundingBox"]["min"]          = {bbCubic.min.x, bbCubic.min.y, bbCubic.min.z};
    metadata["boundingBox"]["max"]          = {bbCubic.max.x, bbCubic.max.y, bbCubic.max.z};
    metadata["encoding"]                    = POTREE_DATA_ENCODING;

    // POSITION attribute
    metadata["attributes"][0]["name"]        = POSITION_NAME;
    metadata["attributes"][0]["description"] = "";
    metadata["attributes"][0]["size"]        = POSITION_SIZE;
    metadata["attributes"][0]["numElements"] = POSITION_ELEMENTS;
    metadata["attributes"][0]["elementSize"] = POSITION_ELEMENT_SIZE;
    metadata["attributes"][0]["type"]        = POSITION_TYPE;
    metadata["attributes"][0]["min"]         = {bbCubic.min.x, bbCubic.min.y, bbCubic.min.z};
    metadata["attributes"][0]["max"]         = {bbCubic.max.x, bbCubic.max.y, bbCubic.max.z};

    // COLOR attribute
    metadata["attributes"][1]["name"]        = COLOR_NAME;
    metadata["attributes"][1]["description"] = "";
    metadata["attributes"][1]["size"]        = COLOR_SIZE;
    metadata["attributes"][1]["numElements"] = COLOR_ELEMENTS;
    metadata["attributes"][1]["elementSize"] = COLOR_ELEMENT_SIZE;
    metadata["attributes"][1]["type"]        = COLOR_TYPE;
    metadata["attributes"][1]["min"]         = {0, 0, 0};
    metadata["attributes"][1]["max"]         = {65024, 65280, 65280};

    std::ofstream file (itsExportFolder + METADATA_FILE_NAME);
    file << std::setw (4) << metadata;
    file.close ();
}
