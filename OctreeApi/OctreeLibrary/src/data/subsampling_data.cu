#include "hip/hip_runtime.h"
#include "subsampling_data.cuh"


const std::unique_ptr<uint32_t[]>& SubsamplingData::getLutHost (uint32_t sparseIndex)
{
    if (itsLutHost.find (sparseIndex) == itsLutHost.end ())
    {
        itsLutHost[sparseIndex] = itsLutDevice[sparseIndex]->toHost ();
    }
    return itsLutHost[sparseIndex];
}

const std::unique_ptr<uint64_t[]>& SubsamplingData::getAvgHost (uint32_t sparseIndex)
{
    if (itsAvgHost.find (sparseIndex) == itsAvgHost.end ())
    {
        itsAvgHost[sparseIndex] = itsAvgDevice[sparseIndex]->toHost ();
    }
    return itsAvgHost[sparseIndex];
}

SubsamplingData::SubsamplingData (uint32_t subsamplingGrid) : itsLastParent (-1)
{
    itsGridCellAmount = static_cast<uint32_t> (pow (subsamplingGrid, 3.f));
}

uint32_t SubsamplingData::getGridCellAmount ()
{
    return itsGridCellAmount;
}

void SubsamplingData::setActiveParent (uint32_t parentNode)
{
    itsLastParent = static_cast<int> (parentNode);
}
int SubsamplingData::getLastParent ()
{
    return itsLastParent;
}
