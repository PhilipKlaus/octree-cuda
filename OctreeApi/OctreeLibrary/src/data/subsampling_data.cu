#include "hip/hip_runtime.h"
#include "random_initialization.cuh"
#include "subsampling_data.cuh"


const std::unique_ptr<uint32_t[]>& SubsamplingData::getLutHost (uint32_t sparseIndex)
{
    if (itsLutHost.find (sparseIndex) == itsLutHost.end ())
    {
        itsLutHost[sparseIndex] = itsLutDevice[sparseIndex]->toHost ();
    }
    return itsLutHost[sparseIndex];
}

const std::unique_ptr<uint64_t[]>& SubsamplingData::getAvgHost (uint32_t sparseIndex)
{
    if (itsAvgHost.find (sparseIndex) == itsAvgHost.end ())
    {
        itsAvgHost[sparseIndex] = itsAvgDevice[sparseIndex]->toHost ();
    }
    return itsAvgHost[sparseIndex];
}

void SubsamplingData::copyToHost ()
{
    itsOutputHost       = itsOutput->toHost ();
}

SubsamplingData::SubsamplingData (uint32_t estimatedPoints, uint32_t subsamplingGrid) : itsLinearCounter (0), itsLastParent(-1)
{
    itsOutput = createGpuOutputData (estimatedPoints, "output");
    itsOutput->memset (0);

    itsGridCellAmount   = static_cast<uint32_t> (pow (subsamplingGrid, 3.f));
    itsCountingGrid     = createGpuU32 (itsGridCellAmount, "pointCountGrid");
    itsAveragingGrid    = createGpuAveraging (itsGridCellAmount, "averagingGrid");
    itsDenseToSparseLut = createGpuI32 (itsGridCellAmount, "denseToSpareLUTSubsamples");
    itsRandomStates     = createGpuRandom (1024, "randomStates");
    itsRandomIndices    = createGpuU32 (itsGridCellAmount, "randomIndices");

    itsCountingGrid->memset (0);
    itsDenseToSparseLut->memset (-1);

    executeKernel (
            subsampling::kernelInitRandoms,
            1024u,
            "kernelInitRandoms",
            std::time (nullptr),
            itsRandomStates->devicePointer (),
            1024);
}

uint32_t SubsamplingData::addLinearLutEntry (uint32_t sparseIdx)
{
    itsLinearLut[sparseIdx] = itsLinearCounter;
    return itsLinearCounter++;
}

uint32_t SubsamplingData::getLinearIdx (uint32_t sparseIndex)
{
    return itsLinearLut[sparseIndex];
}

OutputData* SubsamplingData::getOutputDevice ()
{
    return itsOutput->devicePointer ();
}

OutputData* SubsamplingData::getOutputHost ()
{
    return itsOutputHost.get ();
}

uint32_t* SubsamplingData::getCountingGrid_d ()
{
    return itsCountingGrid->devicePointer ();
}

uint64_t* SubsamplingData::getAverageingGrid_d ()
{
    return itsAveragingGrid->devicePointer ();
}

int32_t* SubsamplingData::getDenseToSparseLut_d ()
{
    return itsDenseToSparseLut->devicePointer ();
}

hiprandState_t* SubsamplingData::getRandomStates_d ()
{
    return itsRandomStates->devicePointer ();
}
uint32_t* SubsamplingData::getRandomIndices_d ()
{
    return itsRandomIndices->devicePointer ();
}

uint32_t SubsamplingData::getGridCellAmount ()
{
    return itsGridCellAmount;
}

void SubsamplingData::setActiveParent (uint32_t parentNode)
{
    itsLastParent = static_cast<int>(parentNode);
}
int SubsamplingData::getLastParent ()
{
    return itsLastParent;
}
