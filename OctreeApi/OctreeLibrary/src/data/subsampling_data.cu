#include "hip/hip_runtime.h"
#include "random_initialization.cuh"
#include "subsampling_data.cuh"


const std::unique_ptr<uint32_t[]>& SubsamplingData::getLutHost (uint32_t sparseIndex)
{
    if (itsLutHost.find (sparseIndex) == itsLutHost.end ())
    {
        itsLutHost[sparseIndex] = itsLutDevice[sparseIndex]->toHost ();
    }
    return itsLutHost[sparseIndex];
}

const std::unique_ptr<uint64_t[]>& SubsamplingData::getAvgHost (uint32_t sparseIndex)
{
    if (itsAvgHost.find (sparseIndex) == itsAvgHost.end ())
    {
        itsAvgHost[sparseIndex] = itsAvgDevice[sparseIndex]->toHost ();
    }
    return itsAvgHost[sparseIndex];
}

void SubsamplingData::copyToHost ()
{
    itsOutputHost       = itsOutput->toHost ();
}

SubsamplingData::SubsamplingData (uint32_t estimatedPoints, uint32_t subsamplingGrid) : itsLastParent(-1)
{
    itsOutput = createGpuOutputData (estimatedPoints, "output");
    itsOutput->memset (0);

    itsGridCellAmount   = static_cast<uint32_t> (pow (subsamplingGrid, 3.f));
    itsAveragingGrid    = createGpuAveraging (itsGridCellAmount, "averagingGrid");
    itsRandomStates     = createGpuRandom (1024, "randomStates");
    itsRandomIndices    = createGpuU32 (itsGridCellAmount, "randomIndices");

    itsAveragingGrid->memset(0);

    executeKernel (
            subsampling::kernelInitRandoms,
            1024u,
            "kernelInitRandoms",
            std::time (nullptr),
            itsRandomStates->devicePointer (),
            1024);
}

OutputData* SubsamplingData::getOutputDevice ()
{
    return itsOutput->devicePointer ();
}

OutputData* SubsamplingData::getOutputHost ()
{
    return itsOutputHost.get ();
}

uint64_t* SubsamplingData::getAverageingGrid_d ()
{
    return itsAveragingGrid->devicePointer ();
}

hiprandState_t* SubsamplingData::getRandomStates_d ()
{
    return itsRandomStates->devicePointer ();
}
uint32_t* SubsamplingData::getRandomIndices_d ()
{
    return itsRandomIndices->devicePointer ();
}

uint32_t SubsamplingData::getGridCellAmount ()
{
    return itsGridCellAmount;
}

void SubsamplingData::setActiveParent (uint32_t parentNode)
{
    itsLastParent = static_cast<int>(parentNode);
}
int SubsamplingData::getLastParent ()
{
    return itsLastParent;
}
