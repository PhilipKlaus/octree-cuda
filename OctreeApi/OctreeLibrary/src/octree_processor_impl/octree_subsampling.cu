#include "hip/hip_runtime.h"
#include "kernel_executor.cuh"
#include "octree_processor_impl.cuh"
#include "random_subsampling.cuh"
#include "subsample_evaluating.cuh"
#include "time_tracker.cuh"


void OctreeProcessor::OctreeProcessorImpl::performSubsampling ()
{
    auto h_sparseToDenseLUT = itsSparseToDenseLUT->toHost ();

    itsDenseToSparseLUT->memset (-1);
    itsCountingGrid->memset (0);

    itsOctree->updateNodeStatistics ();

    randomSubsampling (h_sparseToDenseLUT, itsOctree->getRootIndex (), itsOctree->getMetadata ().depth);
    hipDeviceSynchronize ();
}


void OctreeProcessor::OctreeProcessorImpl::randomSubsampling (
        const unique_ptr<int[]>& h_sparseToDenseLUT, uint32_t sparseVoxelIndex, uint32_t level)
{
    auto& cloudMetadata = itsCloud->getMetadata ();
    auto& node          = itsOctree->getNode (sparseVoxelIndex);

    // Depth first traversal
    for (int childIndex : node.childrenChunks)
    {
        if (childIndex != -1)
        {
            randomSubsampling (h_sparseToDenseLUT, childIndex, level - 1);
        }
    }

    // Now we can assure that all direct children have subsamples
    if (node.isParent)
    {
        // Prepare and update the SubsampleConfig on the GPU
        SubsampleSet subsampleSet{};
        prepareSubsampleConfig (subsampleSet, sparseVoxelIndex);

        // Parent bounding box calculation
        PointCloudMetadata metadata = cloudMetadata;
        auto denseVoxelIndex        = h_sparseToDenseLUT[sparseVoxelIndex];
        calculateVoxelBB (metadata, denseVoxelIndex, level);

        // ToDo: Find more sprecise amount of threads
        KernelStructs::Cloud cloud = {
                itsCloud->getCloudDevice (),
                0,
                metadata.pointDataStride,
                {
                        1.0 / metadata.scale.x,
                        1.0 / metadata.scale.y,
                        1.0 / metadata.scale.z,
                }};
        KernelStructs::Gridding gridding = {
                itsSubsampleMetadata.subsamplingGrid, metadata.cubicSize (), metadata.bbCubic.min};

        Kernel::calcNodeByteOffset (
                {metadata.cloudType, 1, "kernelCalcNodeByteOffset"},
                itsOctree->getDevice (),
                sparseVoxelIndex,
                getLastParent (),
                itsTmpCounting->devicePointer ());

        setActiveParent (sparseVoxelIndex);

        // Evaluate how many points fall in each cell
        Kernel::evaluateSubsamples (
                {metadata.cloudType, itsOctree->getNodeStatistics ().maxPointsPerNode * 8, "kernelEvaluateSubsamples"},
                itsCloud->getOutputBuffer_d (),
                subsampleSet,
                itsCountingGrid->devicePointer (),
                itsOctree->getDevice (),
                itsAveragingGrid->devicePointer (),
                itsDenseToSparseLUT->devicePointer (),
                itsPointLut->devicePointer (),
                cloud,
                gridding,
                sparseVoxelIndex);

        // Prepare one random point index per cell
        auto threads = static_cast<uint32_t> (pow (itsSubsampleMetadata.subsamplingGrid, 3.f));

        executeKernel (
                subsampling::kernelGenerateRandoms,
                threads,
                "kernelGenerateRandoms",
                itsRandomStates->devicePointer (),
                itsRandomIndices->devicePointer (),
                itsDenseToSparseLUT->devicePointer (),
                itsCountingGrid->devicePointer (),
                threads);

        // Distribute the subsampled points in parallel for all child nodes
        Kernel::randomPointSubsampling (
                {metadata.cloudType,
                 itsOctree->getNodeStatistics ().maxPointsPerNode * 8,
                 "kernelRandomPointSubsample"},
                itsCloud->getOutputBuffer_d (),
                subsampleSet,
                itsCountingGrid->devicePointer (),
                itsAveragingGrid->devicePointer (),
                itsDenseToSparseLUT->devicePointer (),
                cloud,
                gridding,
                itsRandomIndices->devicePointer (),
                itsPointLut->devicePointer (),
                itsOctree->getDevice (),
                sparseVoxelIndex);
    }
}


void OctreeProcessor::OctreeProcessorImpl::prepareSubsampleConfig (SubsampleSet& subsampleSet, uint32_t parentIndex)
{
    auto* config = (SubsampleConfig*)(&subsampleSet);
    auto& node   = itsOctree->getNode (parentIndex);
    for (uint8_t i = 0; i < 8; ++i)
    {
        int childIndex      = node.childrenChunks[i];
        config[i].sparseIdx = childIndex;
        if (childIndex != -1)
        {
            Chunk child               = itsOctree->getNode (childIndex);
            config[i].isParent        = child.isParent;
            config[i].leafPointAmount = child.pointCount;
            config[i].leafDataIdx     = child.chunkDataIndex;
        }
        else
        {
            config[i].isParent = false;
        }
    }
}
