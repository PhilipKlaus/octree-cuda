#include "hip/hip_runtime.h"
#include "kernel_executor.cuh"
#include "octree_processor_impl.cuh"
#include "random_subsampling.cuh"
#include "subsample_evaluating.cuh"
#include "time_tracker.cuh"


void OctreeProcessor::OctreeProcessorImpl::performSubsampling ()
{
    auto h_sparseToDenseLUT = itsSparseToDenseLUT->toHost ();

    itsDenseToSparseLUT->memset (-1);
    itsCountingGrid->memset (0);
    auto timing = Timing::TimeTracker::start ();
    itsOctree->updateNodeStatistics ();
    Timing::TimeTracker::stop (timing, "Update octree statistics (CPU)", Timing::Time::PROCESS);
    randomSubsampling (h_sparseToDenseLUT, itsOctree->getRootIndex (), itsOctree->getMetadata ().depth);
    hipDeviceSynchronize ();
}


void OctreeProcessor::OctreeProcessorImpl::randomSubsampling (
        const unique_ptr<int[]>& h_sparseToDenseLUT, uint32_t sparseVoxelIndex, uint32_t level)
{
    auto& cloudMetadata = itsCloud->getMetadata ();
    auto& node          = itsOctree->getNode (sparseVoxelIndex);

    // Depth first traversal
    for (int childIndex : node.childrenChunks)
    {
        if (childIndex != -1)
        {
            randomSubsampling (h_sparseToDenseLUT, childIndex, level - 1);
        }
    }

    // Now we can assure that all direct children have subsamples
    if (node.isParent)
    {
        // Parent bounding box calculation
        PointCloudMetadata metadata = cloudMetadata;
        auto denseVoxelIndex        = h_sparseToDenseLUT[sparseVoxelIndex];
        calculateVoxelBB (metadata, denseVoxelIndex, level);

        // ToDo: Find more sprecise amount of threads
        KernelStructs::Cloud cloud = {
                itsCloud->getCloudDevice (),
                0,
                metadata.pointDataStride,
                {
                        1.0 / metadata.scale.x,
                        1.0 / metadata.scale.y,
                        1.0 / metadata.scale.z,
                }};
        KernelStructs::Gridding gridding = {
                itsSubsampleMetadata.subsamplingGrid, metadata.cubicSize (), metadata.bbCubic.min};

        Kernel::calcNodeByteOffset (
                {metadata.cloudType, 1, "kernelCalcNodeByteOffset"},
                itsOctree->getDevice (),
                sparseVoxelIndex,
                getLastParent (),
                itsTmpCounting->devicePointer ());

        setActiveParent (sparseVoxelIndex);

        // Evaluate how many points fall in each cell
        Kernel::evaluateSubsamples (
                {metadata.cloudType, itsOctree->getNodeStatistics ().maxPointsPerNode * 8, "kernelEvaluateSubsamples"},
                itsCloud->getOutputBuffer_d (),
                itsCountingGrid->devicePointer (),
                itsOctree->getDevice (),
                itsAveragingGrid->devicePointer (),
                itsDenseToSparseLUT->devicePointer (),
                itsPointLut->devicePointer (),
                cloud,
                gridding,
                sparseVoxelIndex);

        // Prepare one random point index per cell
        auto threads = static_cast<uint32_t> (pow (itsSubsampleMetadata.subsamplingGrid, 3.f));

        executeKernel (
                subsampling::kernelGenerateRandoms,
                threads,
                "kernelGenerateRandoms",
                itsRandomStates->devicePointer (),
                itsRandomIndices->devicePointer (),
                itsDenseToSparseLUT->devicePointer (),
                itsCountingGrid->devicePointer (),
                threads);

        // Distribute the subsampled points in parallel for all child nodes
        Kernel::randomPointSubsampling (
                {metadata.cloudType,
                 itsOctree->getNodeStatistics ().maxPointsPerNode * 8,
                 "kernelRandomPointSubsample"},
                itsCloud->getOutputBuffer_d (),
                itsCountingGrid->devicePointer (),
                itsAveragingGrid->devicePointer (),
                itsDenseToSparseLUT->devicePointer (),
                cloud,
                gridding,
                itsRandomIndices->devicePointer (),
                itsPointLut->devicePointer (),
                itsOctree->getDevice (),
                sparseVoxelIndex);
    }
}
