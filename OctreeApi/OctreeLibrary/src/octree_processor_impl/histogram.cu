#include "hip/hip_runtime.h"
//
// Created by KlausP on 13.11.2020.
//

#include "octree_processor_impl.cuh"

void OctreeProcessor::OctreeProcessorImpl::updateOctreeStatistics ()
{
    itsOctree->updateNodeStatistics ();
}


void OctreeProcessor::OctreeProcessorImpl::histogramBinning (
        const shared_ptr<Chunk[]>& h_octreeSparse,
        std::vector<uint32_t>& counts,
        uint32_t min,
        uint32_t binWidth,
        uint32_t nodeIndex) const
{
    Chunk chunk = h_octreeSparse[nodeIndex];

    // Leaf node
    if (!chunk.isParent)
    {
        uint32_t bin = (chunk.pointCount - min) / binWidth;
        ++counts[bin];
    }

    // Parent node
    else
    {
        for (uint32_t i = 0; i < 8; ++i)
        {
            int childIndex = chunk.childrenChunks[i];
            if (childIndex != -1)
            {
                histogramBinning (h_octreeSparse, counts, min, binWidth, chunk.childrenChunks[i]);
            }
        }
    }
}


void OctreeProcessor::OctreeProcessorImpl::exportHistogram (const string& filePath, uint32_t binWidth)
{
    updateOctreeStatistics ();
    auto& statistics = itsOctree->getNodeStatistics ();

    if (binWidth == 0)
    {
        binWidth = static_cast<uint32_t> (
                ceil (3.5f * (statistics.stdevPointsPerLeafNode / pow (statistics.leafNodeAmount, 1.f / 3.f))));
    }
    auto binAmount =
            static_cast<uint32_t> (ceil (statistics.maxPointsPerNode - statistics.minPointsPerNode) / binWidth);
    std::vector<uint32_t> counts;
    for (uint32_t i = 0; i < binAmount; ++i)
    {
        counts.push_back (0);
    }
    histogramBinning (itsOctree->getHost (), counts, statistics.minPointsPerNode, binWidth, itsOctree->getRootIndex ());

    const std::string itsHtmlPart1 = "<html>\n"
                                     "    <head>\n"
                                     "        <script src=\"https://cdn.jsdelivr.net/npm/chart.js@2.8.0\"></script>\n"
                                     "        <style>\n"
                                     "            html, body {\n"
                                     "            margin: 0;\n"
                                     "            height: 100%;\n"
                                     "            }\n"
                                     "            canvas {\n"
                                     "            width: 100%;\n"
                                     "            height: 100%;\n"
                                     "            display: block;\n"
                                     "            }\n"
                                     "        </style>\n"
                                     "    </head>\n"
                                     "    <body>\n"
                                     "        <canvas id=\"myChart\"></canvas>\n"
                                     "        <script>\n"
                                     "            var ctx = document.getElementById('myChart').getContext('2d');\n"
                                     "            var chart = new Chart(ctx, {\n"
                                     "                // The type of chart we want to create\n"
                                     "                type: 'bar',\n"
                                     "\n"
                                     "                // The data for our dataset\n"
                                     "                data:{";

    const std::string itsHtmlPart2 = "               }\n"
                                     "            });\n"
                                     "        </script>\n"
                                     "    </body>\n"
                                     "</html>";

    string labels = "labels:[";
    string data   = "data:[";
    string label  = "'Point Distribution: binWidth(" + to_string (binWidth) + "), mergingThreshold(" +
                   to_string (itsOctree->getMetadata ().mergingThreshold) + "), points(" +
                   to_string (itsCloud->getMetadata ().pointAmount) + ")'";

    for (uint32_t i = 0; i < binAmount; ++i)
    {
        labels +=
                ("'" + to_string (statistics.minPointsPerNode + i * binWidth) + " - " +
                 to_string (statistics.minPointsPerNode + (i + 1) * binWidth) + "'");
        data += to_string (counts[i]);
        if (i < (binAmount - 1))
        {
            labels += ",";
            data += ",";
        }
    }
    labels += "]";
    data += "]";

    std::ofstream htmlData;
    htmlData.open (std::string (filePath), std::ios::out);
    htmlData << itsHtmlPart1;
    htmlData << labels + ", datasets:[{label: " + label +
                        ", backgroundColor: 'rgb(255, 99, 132)', borderColor: 'rgb(255, 99, 132)', " + data + "}]";
    htmlData << itsHtmlPart2;
    htmlData.close ();
};
