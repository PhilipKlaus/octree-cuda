#include "hip/hip_runtime.h"
#include "fp_subsample_evaluation.cuh"
#include "fp_subsampling.cuh"
#include "kernel_executor.cuh"
#include "kernel_helpers.cuh"
#include "kernel_structs.cuh"
#include "octree_processor_impl.cuh"

void OctreeProcessor::OctreeProcessorImpl::firstPointSubsampling (
        uint32_t sparseVoxelIndex, uint32_t level, Vector3<double> nodeBBMin)
{
    auto& cloudMetadata = itsCloud->getMetadata ();
    auto& node          = itsOctree->getNode (sparseVoxelIndex);

    // Calculate actual cell (node) side length
    auto bbDivider = pow (2, itsOctree->getNodeStatistics ().depth - level);
    double side    = (cloudMetadata.bbCubic.max.x - cloudMetadata.bbCubic.min.x) / bbDivider;

    // Depth first traversal
    uint8_t tmpIndex = 0;
    for (int childIndex : node.childNodes)
    {
        if (childIndex != -1)
        {
            auto childBBSide           = side / 2.0;
            Vector3<double> childBBMin = nodeBBMin;
            tools::calculateChildMinBB (childBBMin, nodeBBMin, tmpIndex, childBBSide);
            firstPointSubsampling (childIndex, level - 1, childBBMin);
        }
        ++tmpIndex;
    }

    // Now we can assure that all direct children have subsamples
    if (node.isInternal)
    {
        // Parent bounding box calculation
        PointCloudInfo metadata = cloudMetadata;
        metadata.bbCubic.min    = nodeBBMin;
        metadata.bbCubic.max.x  = metadata.bbCubic.min.x + side;
        metadata.bbCubic.max.y  = metadata.bbCubic.min.y + side;
        metadata.bbCubic.max.z  = metadata.bbCubic.min.z + side;
        metadata.cloudOffset    = metadata.bbCubic.min;


        // ToDo: Find more sprecise amount of threads
        KernelStructs::Cloud cloud = {
                itsCloud->getCloudDevice (),
                0,
                metadata.pointDataStride,
                {
                        1.0 / metadata.scale.x,
                        1.0 / metadata.scale.y,
                        1.0 / metadata.scale.z,
                }};
        KernelStructs::Gridding gridding = {
                itsProcessingInfo.subsamplingGrid, metadata.cubicSize (), metadata.bbCubic.min};

        // No color averagin is performed -> directly subsample points
        if (!itsProcessingInfo.useIntraCellAvg && !itsProcessingInfo.useInterCellAvg)
        {
            Kernel::fp::subsampleNotAveraged (
                    {metadata.cloudType,
                     itsOctree->getNodeStatistics ().maxPointsPerNode * 8,
                     "kernelRandomPointSubsample"},
                    itsCloud->getOutputBuffer_d (),
                    itsCountingGrid->devicePointer (),
                    itsDenseToSparseLUT->devicePointer (),
                    cloud,
                    gridding,
                    cloudMetadata.bbCubic,
                    itsPointLut->devicePointer (),
                    itsOctree->getDevice (),
                    sparseVoxelIndex,
                    getLastParent (),
                    itsTmpCounting->devicePointer ());

            auto gridCellAmount = static_cast<uint32_t> (pow (itsProcessingInfo.subsamplingGrid, 3.f));

            executeKernel (
                    tools::kernelMemset1D<uint32_t>,
                    gridCellAmount,
                    "kernelMemset1D",
                    itsCountingGrid->devicePointer (),
                    0,
                    gridCellAmount);
        }
        else
        {
            // Intra-cell averaging
            if (itsProcessingInfo.useIntraCellAvg)
            {
                Kernel::fp::evaluateSubsamplesIntra (
                        {metadata.cloudType,
                         itsOctree->getNodeStatistics ().maxPointsPerNode * 8,
                         "kernelEvaluateSubsamplesIntra"},
                        itsCloud->getOutputBuffer_d (),
                        itsOctree->getDevice (),
                        itsAveragingGrid->devicePointer (),
                        itsDenseToSparseLUT->devicePointer (),
                        itsPointLut->devicePointer (),
                        cloud,
                        gridding,
                        sparseVoxelIndex,
                        getLastParent (),
                        itsTmpCounting->devicePointer ());
            }
            // Inter-cell averaging
            else
            {
                Kernel::fp::evaluateSubsamplesInter (
                        {metadata.cloudType,
                         itsOctree->getNodeStatistics ().maxPointsPerNode * 8,
                         "kernelEvaluateSubsamplesInter"},
                        itsCountingGrid->devicePointer (),
                        itsOctree->getDevice (),
                        itsDenseToSparseLUT->devicePointer (),
                        itsPointLut->devicePointer (),
                        cloud,
                        gridding,
                        sparseVoxelIndex,
                        getLastParent (),
                        itsTmpCounting->devicePointer ());

                // Inter-Cell: Accumulate colors from neighbouring cells
                Kernel::fp::interCellAvg (
                        {metadata.cloudType,
                         itsOctree->getNodeStatistics ().maxPointsPerNode * 8,
                         "kernelInterCellAveraging"},
                        itsCloud->getOutputBuffer_d (),
                        itsCountingGrid->devicePointer (),
                        itsOctree->getDevice (),
                        itsAveragingGrid->devicePointer (),
                        itsDenseToSparseLUT->devicePointer (),
                        itsPointLut->devicePointer (),
                        cloud,
                        gridding,
                        sparseVoxelIndex);
            }

            // Finally subsample the points
            Kernel::fp::subsampleAveraged (
                    {metadata.cloudType,
                     itsOctree->getNodeStatistics ().maxPointsPerNode * 8,
                     "kernelRandomPointSubsample"},
                    itsCloud->getOutputBuffer_d (),
                    itsCountingGrid->devicePointer (),
                    itsAveragingGrid->devicePointer (),
                    itsDenseToSparseLUT->devicePointer (),
                    cloud,
                    gridding,
                    cloudMetadata.bbCubic,
                    itsPointLut->devicePointer (),
                    itsOctree->getDevice (),
                    sparseVoxelIndex);
        }

        setActiveParent (sparseVoxelIndex);
    }
}
