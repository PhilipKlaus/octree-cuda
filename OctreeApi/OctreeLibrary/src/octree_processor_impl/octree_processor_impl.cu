#include "hip/hip_runtime.h"
//
// Created by KlausP on 04.11.2020.
//

#include "octree_processor_impl.cuh"
#include "ply_exporter.cuh"
#include "potree_exporter.cuh"
#include "time_tracker.cuh"
#include "tools.cuh"

OctreeProcessor::OctreeProcessorImpl::OctreeProcessorImpl (
        uint8_t* pointCloud,
        uint32_t chunkingGrid,
        uint32_t mergingThreshold,
        PointCloudMetadata cloudMetadata,
        SubsampleMetadata subsamplingMetadata)
{
    itsOctreeData = std::make_unique<Octree> (chunkingGrid);

    // ToDo: Move itsMeatadata to OctreeData
    // Initialize metadata
    itsMetadata                  = {};
    itsMetadata.depth            = itsOctreeData->getDepth ();
    itsMetadata.nodeAmountDense  = itsOctreeData->getOverallNodes ();
    itsMetadata.chunkingGrid     = chunkingGrid;
    itsMetadata.mergingThreshold = mergingThreshold;
    itsSubsampleMetadata         = subsamplingMetadata;

    if (cloudMetadata.memoryType == CLOUD_HOST)
    {
        itsCloud = std::make_unique<PointCloudHost> (pointCloud, cloudMetadata);
    }
    else
    {
        itsCloud = std::make_unique<PointCloudDevice> (pointCloud, cloudMetadata);
    }

    //-----------------------------
    // Create GPU data for chunking
    //-----------------------------

    auto start = std::chrono::high_resolution_clock::now ();

    // Allocate the dense point count
    itsDensePointCountPerVoxel = createGpuU32 (itsMetadata.nodeAmountDense, "DensePointCountPerVoxel");
    itsDensePointCountPerVoxel->memset (0);

    // Allocate the conversion LUT from dense to sparse
    itsDenseToSparseLUT = createGpuI32 (itsMetadata.nodeAmountDense, "DenseToSparseLUT");
    itsDenseToSparseLUT->memset (-1);

    // Allocate the temporary sparseIndexCounter
    itsTmpCounting = createGpuU32 (1, "nodeAmountSparse");
    itsTmpCounting->memset (0);

    itsLeafLut = createGpuU32 (cloudMetadata.pointAmount, "Data LUT");

    itsSubsamples = std::make_shared<SubsamplingData> (
            itsCloud->getMetadata ().pointAmount * 2.2, itsSubsampleMetadata.subsamplingGrid);

    auto finish                           = std::chrono::high_resolution_clock::now ();
    std::chrono::duration<double> elapsed = finish - start;
    spdlog::info("Allocating GPU data structures took: {} [s]", elapsed.count());
}

void OctreeProcessor::OctreeProcessorImpl::calculateVoxelBB (
        PointCloudMetadata& metadata, uint32_t denseVoxelIndex, uint32_t level)
{
    Vector3<uint32_t> coords = {};

    // 1. Calculate coordinates of voxel within the actual level
    auto indexInLevel = denseVoxelIndex - itsOctreeData->getNodeOffset (level);
    tools::mapFromDenseIdxToDenseCoordinates (coords, indexInLevel, itsOctreeData->getGridSize (level));

    // 2. Calculate the bounding box for the actual voxel
    // ToDo: Include scale and offset!!!
    auto& cloudMeta = itsCloud->getMetadata ();
    double min      = cloudMeta.bbCubic.min.x;
    double max      = cloudMeta.bbCubic.max.x;
    double side     = max - min;
    auto cubicWidth = side / itsOctreeData->getGridSize (level);

    metadata.bbCubic.min.x = cloudMeta.bbCubic.min.x + coords.x * cubicWidth;
    metadata.bbCubic.min.y = cloudMeta.bbCubic.min.y + coords.y * cubicWidth;
    metadata.bbCubic.min.z = cloudMeta.bbCubic.min.z + coords.z * cubicWidth;
    metadata.bbCubic.max.x = metadata.bbCubic.min.x + cubicWidth;
    metadata.bbCubic.max.y = metadata.bbCubic.min.y + cubicWidth;
    metadata.bbCubic.max.z = metadata.bbCubic.min.z + cubicWidth;
    metadata.cloudOffset   = metadata.bbCubic.min;
}

void OctreeProcessor::OctreeProcessorImpl::exportPotree (const string& folderPath)
{
    itsSubsamples->copyToHost ();

    if (itsCloud->getMetadata ().cloudType == CLOUD_FLOAT_UINT8_T)
    {
        PotreeExporter<float, uint8_t> potreeExporter (
                itsCloud,
                itsOctreeData->getHost (),
                itsLeafLut,
                itsSubsamples,
                itsMetadata,
                itsCloud->getMetadata (),
                itsSubsampleMetadata);
        potreeExporter.exportOctree (folderPath);
    }
    else
    {
        PotreeExporter<double, uint8_t> potreeExporter (
                itsCloud,
                itsOctreeData->getHost (),
                itsLeafLut,
                itsSubsamples,
                itsMetadata,
                itsCloud->getMetadata (),
                itsSubsampleMetadata);
        potreeExporter.exportOctree (folderPath);
    }
}

void OctreeProcessor::OctreeProcessorImpl::exportPlyNodes (const string& folderPath)
{
    itsSubsamples->copyToHost ();

    auto start = std::chrono::high_resolution_clock::now ();

    if (itsCloud->getMetadata ().cloudType == CLOUD_FLOAT_UINT8_T)
    {
        PlyExporter<float, uint8_t> plyExporter (
                itsCloud,
                itsOctreeData->getHost (),
                itsLeafLut,
                itsSubsamples,
                itsMetadata,
                itsCloud->getMetadata (),
                itsSubsampleMetadata);
        plyExporter.exportOctree (folderPath);
    }
    else
    {
        PotreeExporter<double, uint8_t> plyExporter (
                itsCloud,
                itsOctreeData->getHost (),
                itsLeafLut,
                itsSubsamples,
                itsMetadata,
                itsCloud->getMetadata (),
                itsSubsampleMetadata);
        plyExporter.exportOctree (folderPath);
    }
}
