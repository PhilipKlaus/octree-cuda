#include "hip/hip_runtime.h"
//
// Created by KlausP on 04.11.2020.
//

#include "octree_processor_impl.cuh"
#include "ply_exporter.cuh"
#include "potree_exporter.cuh"
#include "time_tracker.cuh"
#include "tools.cuh"

OctreeProcessor::OctreeProcessorImpl::OctreeProcessorImpl (
        uint8_t* pointCloud,
        uint32_t chunkingGrid,
        uint32_t mergingThreshold,
        PointCloudMetadata cloudMetadata,
        SubsampleMetadata subsamplingMetadata)
{
    itsOctreeData = std::make_unique<Octree> (chunkingGrid);

    // ToDo: Move itsMeatadata to OctreeData
    // Initialize metadata
    itsMetadata                  = {};
    itsMetadata.depth            = itsOctreeData->getDepth ();
    itsMetadata.nodeAmountDense  = itsOctreeData->getOverallNodes ();
    itsMetadata.chunkingGrid     = chunkingGrid;
    itsMetadata.mergingThreshold = mergingThreshold;
    itsSubsampleMetadata         = subsamplingMetadata;

    if (cloudMetadata.memoryType == CLOUD_HOST)
    {
        itsCloud = std::make_unique<PointCloudHost> (pointCloud, cloudMetadata);
    }
    else
    {
        itsCloud = std::make_unique<PointCloudDevice> (pointCloud, cloudMetadata);
    }

    //-----------------------------
    // Create GPU data for chunking
    //-----------------------------

    auto timing = Timing::TimeTracker::start ();

    // Allocate the dense point count
    itsCountingGrid = createGpuU32 (itsMetadata.nodeAmountDense, "countingGrid");
    itsCountingGrid->memset (0);

    // Allocate the conversion LUT from dense to sparse
    itsDenseToSparseLUT = createGpuI32 (itsMetadata.nodeAmountDense, "denseToSparseLut");
    itsDenseToSparseLUT->memset (-1);

    // Allocate the temporary sparseIndexCounter
    itsTmpCounting = createGpuU32 (1, "tmpCounting");
    itsTmpCounting->memset (0);

    itsSubsamples       = std::make_shared<SubsamplingData> (itsSubsampleMetadata.subsamplingGrid);

    auto expectedPoints = static_cast<uint32_t> (itsCloud->getMetadata ().pointAmount * 2.1);
    itsPointLut = createGpuOutputData (expectedPoints, "pointLUT");
    itsPointLut->memset (0);

    Timing::TimeTracker::stop (timing, "Preparing GPU data", Timing::Time::PROCESS);
}

void OctreeProcessor::OctreeProcessorImpl::calculateVoxelBB (
        PointCloudMetadata& metadata, uint32_t denseVoxelIndex, uint32_t level)
{
    Vector3<uint32_t> coords = {};

    // 1. Calculate coordinates of voxel within the actual level
    auto indexInLevel = denseVoxelIndex - itsOctreeData->getNodeOffset (level);
    tools::mapFromDenseIdxToDenseCoordinates (coords, indexInLevel, itsOctreeData->getGridSize (level));

    // 2. Calculate the bounding box for the actual voxel
    // ToDo: Include scale and offset!!!
    auto& cloudMeta = itsCloud->getMetadata ();
    double min      = cloudMeta.bbCubic.min.x;
    double max      = cloudMeta.bbCubic.max.x;
    double side     = max - min;
    auto cubicWidth = side / itsOctreeData->getGridSize (level);

    metadata.bbCubic.min.x = cloudMeta.bbCubic.min.x + coords.x * cubicWidth;
    metadata.bbCubic.min.y = cloudMeta.bbCubic.min.y + coords.y * cubicWidth;
    metadata.bbCubic.min.z = cloudMeta.bbCubic.min.z + coords.z * cubicWidth;
    metadata.bbCubic.max.x = metadata.bbCubic.min.x + cubicWidth;
    metadata.bbCubic.max.y = metadata.bbCubic.min.y + cubicWidth;
    metadata.bbCubic.max.z = metadata.bbCubic.min.z + cubicWidth;
    metadata.cloudOffset   = metadata.bbCubic.min;
}

void OctreeProcessor::OctreeProcessorImpl::exportPotree (const string& folderPath)
{
    itsOctreeData->copyToHost();

    if (itsCloud->getMetadata ().cloudType == CLOUD_FLOAT_UINT8_T)
    {
        PotreeExporter<float, uint8_t> potreeExporter (
                itsCloud,
                itsOctreeData->getHost (),
                itsSubsamples,
                itsMetadata,
                itsCloud->getMetadata (),
                itsSubsampleMetadata);
        potreeExporter.exportOctree (folderPath);
    }
    else
    {
        PotreeExporter<double, uint8_t> potreeExporter (
                itsCloud,
                itsOctreeData->getHost (),
                itsSubsamples,
                itsMetadata,
                itsCloud->getMetadata (),
                itsSubsampleMetadata);
        potreeExporter.exportOctree (folderPath);
    }
}

void OctreeProcessor::OctreeProcessorImpl::exportPlyNodes (const string& folderPath)
{
    itsOctreeData->copyToHost();

    if (itsCloud->getMetadata ().cloudType == CLOUD_FLOAT_UINT8_T)
    {
        PlyExporter<float, uint8_t> plyExporter (
                itsCloud,
                itsOctreeData->getHost (),
                itsSubsamples,
                itsMetadata,
                itsCloud->getMetadata (),
                itsSubsampleMetadata);
        plyExporter.exportOctree (folderPath);
    }
    else
    {
        PotreeExporter<double, uint8_t> plyExporter (
                itsCloud,
                itsOctreeData->getHost (),
                itsSubsamples,
                itsMetadata,
                itsCloud->getMetadata (),
                itsSubsampleMetadata);
        plyExporter.exportOctree (folderPath);
    }
}
