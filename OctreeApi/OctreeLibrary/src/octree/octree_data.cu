#include "hip/hip_runtime.h"
#include "octree_data.cuh"
#include "tools.cuh"

OctreeData::OctreeData (uint32_t chunkingGrid) : itsDepth (0), itsChunkingGrid (chunkingGrid), itsNodeAmountDense (0)
{
    initialize ();
}

void OctreeData::initialize ()
{
    itsDepth = tools::getOctreeLevel (itsChunkingGrid);

    for (uint32_t gridSize = itsChunkingGrid; gridSize > 0; gridSize >>= 1)
    {
        itsGridSizePerLevel.push_back (gridSize);
        itsNodeOffsetperLevel.push_back (itsNodeAmountDense);
        itsNodesPerLevel.push_back (static_cast<uint32_t> (pow (gridSize, 3)));
        itsNodeAmountDense += static_cast<uint32_t> (pow (gridSize, 3));
    }
}
uint8_t OctreeData::getDepth ()
{
    return itsDepth;
}
uint32_t OctreeData::getNodes (uint8_t level)
{
    return itsNodesPerLevel[level];
}
uint32_t OctreeData::getGridSize (uint8_t level)
{
    return itsGridSizePerLevel[level];
}
uint32_t OctreeData::getNodeOffset (uint8_t level)
{
    return itsNodeOffsetperLevel[level];
}
uint32_t OctreeData::getOverallNodes ()
{
    return itsNodeAmountDense;
}
void OctreeData::createOctree (uint32_t nodeAmountSparse)
{
    itsOctree = createGpuOctree (nodeAmountSparse, "octreeSparse");
}
void OctreeData::copyToHost ()
{
    auto start = std::chrono::high_resolution_clock::now ();
    itsOctreeHost = itsOctree->toHost();
    auto finish                           = std::chrono::high_resolution_clock::now ();
    std::chrono::duration<double> elapsed = finish - start;
    spdlog::info("Copied octree from device to host in: {}s", elapsed.count());
}
const std::shared_ptr<Chunk[]>& OctreeData::getHost ()
{
    if(!itsOctreeHost) {
        copyToHost();
    }
    return itsOctreeHost;
}
Chunk* OctreeData::getDevice ()
{
    return itsOctree->devicePointer();
}
