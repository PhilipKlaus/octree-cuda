#include "hip/hip_runtime.h"
#include <chunking.cuh>
#include <tools.cuh>
#include <timing.cuh>


__global__ void chunking::kernelInitialPointCounting(
        Vector3 *cloud,
        uint32_t *densePointCount,
        int *denseToSparseLUT,
        uint32_t *sparseIndexCounter,
        PointCloudMetadata metadata,
        uint32_t gridSideLength
) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= metadata.pointAmount) {
        return;
    }
    Vector3 point = cloud[index];

    // 1. Calculate the index within the dense grid
    auto denseVoxelIndex = tools::calculateGridIndex(point, metadata, gridSideLength);

    // 2. Accumulate the counter within the dense cell
    auto oldIndex = atomicAdd((densePointCount + denseVoxelIndex), 1);

    // 3. If the thread is the first one accumulating the counter within the cell -> update the denseToSparseLUT
    if(oldIndex == 0) {
        auto sparseVoxelIndex = atomicAdd(sparseIndexCounter, 1);
        denseToSparseLUT[denseVoxelIndex] = sparseVoxelIndex;
    }
}


float chunking::initialPointCounting(
        unique_ptr<CudaArray<Vector3>> &cloud,
        unique_ptr<CudaArray<uint32_t>> &densePointCount,
        unique_ptr<CudaArray<int>> &denseToSparseLUT,
        unique_ptr<CudaArray<uint32_t>> &sparseIndexCounter,
        PointCloudMetadata metadata,
        uint32_t gridSideLength
) {
    // Calculate kernel dimensions
    dim3 grid, block;
    tools::create1DKernel(block, grid, metadata.pointAmount);

    // Initial point counting
    tools::KernelTimer timer;
    timer.start();
    chunking::kernelInitialPointCounting <<<  grid, block >>> (
            cloud->devicePointer(),
            densePointCount->devicePointer(),
            denseToSparseLUT->devicePointer(),
            sparseIndexCounter->devicePointer(),
            metadata,
            gridSideLength);
    timer.stop();
    gpuErrchk(hipGetLastError());

    spdlog::info("'kernelMapCloudToGrid' took {:f} [ms]", timer.getMilliseconds());
    return timer.getMilliseconds();
}
