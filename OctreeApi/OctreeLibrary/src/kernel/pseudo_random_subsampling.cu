#include "hip/hip_runtime.h"
#include "pseudo_random_subsampling.cuh"
#include "../../include/types.h"
#include <tools.cuh>
#include <timing.cuh>


// Move point indices from old (child LUT) to new (parent LUT)
__global__ void pseudo__random_subsampling::kernelDistributeSubsamples(
        uint8_t *cloud,
        uint32_t *childDataLUT,
        uint32_t childDataLUTStart,
        uint32_t *parentDataLUT,
        uint32_t *countingGrid,
        int *denseToSparseLUT,
        uint32_t *sparseIndexCounter,
        PointCloudMetadata metadata,
        uint32_t gridSideLength,
        uint32_t *randomIndices
) {
    int index = (blockIdx.y * gridDim.x * blockDim.x) + (blockIdx.x * blockDim.x + threadIdx.x);
    if(index >= metadata.pointAmount) {
        return;
    }

    auto *point = reinterpret_cast<Vector3 *>(cloud + childDataLUT[childDataLUTStart + index] * metadata.pointDataStride);


    // 1. Calculate the index within the dense grid of the subsample
    auto denseVoxelIndex = tools::calculateGridIndex(point, metadata, gridSideLength);

    // 2. We are only interested in the last point within a node -> Implicitly reset the countingGrid
    auto oldIndex = atomicSub((countingGrid + denseVoxelIndex), 1);

    // 3. If the thread is the first one ->
    //      3.1 store the child lut table index in the parent lut
    //      3.2 'delete' the point within the child lut by invalidating its index entry

    int sparseIndex = denseToSparseLUT[denseVoxelIndex];

    if(oldIndex == randomIndices[sparseIndex]) {
        parentDataLUT[sparseIndex] = childDataLUT[childDataLUTStart + index];
        childDataLUT[childDataLUTStart + index] = INVALID_INDEX;
    }
    if(oldIndex == 1) {
        // Reset data structures
        denseToSparseLUT[denseVoxelIndex] = -1;
        *sparseIndexCounter = 0;
    }
}

// http://ianfinlayson.net/class/cpsc425/notes/cuda-random
__global__ void pseudo__random_subsampling::kernelInitRandoms(
        unsigned int seed,
        hiprandState_t *states,
        uint32_t nodeAmount) {

    int index = (blockIdx.y * gridDim.x * blockDim.x) + (blockIdx.x * blockDim.x + threadIdx.x);

    if(index >= nodeAmount) {
        return;
    }

    /* we have to initialize the state */
    hiprand_init(
            seed,
            index,
            0,
            &states[index]);
}

__global__ void pseudo__random_subsampling::kernelGenerateRandoms(
        hiprandState_t* states,
        uint32_t *randomIndices,
        const int *denseToSparseLUT,
        const uint32_t *countingGrid,
        uint32_t gridNodes) {

    int index = (blockIdx.y * gridDim.x * blockDim.x) + (blockIdx.x * blockDim.x + threadIdx.x);

    if(index >= gridNodes) {
        return;
    }

    int sparseIndex = denseToSparseLUT[index];

    if(sparseIndex > -1) {
        randomIndices[sparseIndex] = static_cast<uint32_t>(ceil(hiprand_uniform(&states[sparseIndex]) * countingGrid[index]));
    }
}

__global__ void pseudo__random_subsampling::kernelSubsample(
        uint8_t *cloud,
        uint32_t *cloudDataLUT,
        uint32_t dataLUTStartIndex,
        uint32_t *densePointCount,
        int *denseToSparseLUT,
        uint32_t *sparseIndexCounter,
        PointCloudMetadata metadata,
        uint32_t gridSideLength
) {
    int index = (blockIdx.y * gridDim.x * blockDim.x) + (blockIdx.x * blockDim.x + threadIdx.x);
    if(index >= metadata.pointAmount) {
        return;
    }

    //Vector3 point = cloud[cloudDataLUT[dataLUTStartIndex + index]];
    auto *point = reinterpret_cast<Vector3 *>(cloud + cloudDataLUT[dataLUTStartIndex + index] * metadata.pointDataStride);

    // 1. Calculate the index within the dense grid of the subsample
    auto denseVoxelIndex = tools::calculateGridIndex(point, metadata, gridSideLength);

    // 2. We are only interested in the first point within a cell
    auto oldIndex = atomicAdd((densePointCount + denseVoxelIndex), 1);

    // 3. If the thread is the first one -> increase map from the dense grid to the sparse grid
    if(oldIndex == 0) {
        auto sparseVoxelIndex = atomicAdd(sparseIndexCounter, 1);
        denseToSparseLUT[denseVoxelIndex] = sparseVoxelIndex;
    }
}

float pseudo__random_subsampling::distributeSubsamples(
        unique_ptr<CudaArray<uint8_t>> &cloud,
        unique_ptr<CudaArray<uint32_t>> &childDataLUT,
        uint32_t childDataLUTStart,
        unique_ptr<CudaArray<uint32_t>> &parentDataLUT,
        unique_ptr<CudaArray<uint32_t>> &countingGrid,
        unique_ptr<CudaArray<int>> &denseToSparseLUT,
        unique_ptr<CudaArray<uint32_t>> &sparseIndexCounter,
        PointCloudMetadata metadata,
        uint32_t gridSideLength,
        unique_ptr<CudaArray<uint32_t>> &randomIndices
) {
    // Calculate kernel dimensions
    dim3 grid, block;
    tools::create1DKernel(block, grid, metadata.pointAmount);

    // Initial point counting
    tools::KernelTimer timer;
    timer.start();
    kernelDistributeSubsamples <<<  grid, block >>> (
            cloud->devicePointer(),
                    childDataLUT->devicePointer(),
                    childDataLUTStart,
                    parentDataLUT->devicePointer(),
                    countingGrid->devicePointer(),
                    denseToSparseLUT->devicePointer(),
                    sparseIndexCounter->devicePointer(),
                    metadata,
                    gridSideLength,
                    randomIndices->devicePointer());
    timer.stop();
    gpuErrchk(hipGetLastError());

    spdlog::debug("'kernelDistributeSubsamples' took {:f} [ms]", timer.getMilliseconds());
    return timer.getMilliseconds();
}

float pseudo__random_subsampling::generateRandoms(
        const unique_ptr<CudaArray<hiprandState_t>> &states,
        unique_ptr<CudaArray<uint32_t>> &randomIndices,
        const unique_ptr<CudaArray<int>> &denseToSparseLUT,
        const unique_ptr<CudaArray<uint32_t>> &countingGrid,
        uint32_t gridNodes) {

    // Calculate kernel dimensions
    dim3 grid, block;
    tools::create1DKernel(block, grid, gridNodes);

    // Initial point counting
    tools::KernelTimer timer;
    timer.start();
    kernelGenerateRandoms << < grid, block >> > (states->devicePointer(), randomIndices->devicePointer(), denseToSparseLUT->devicePointer(), countingGrid->devicePointer(), gridNodes);
    timer.stop();
    gpuErrchk(hipGetLastError());

    spdlog::debug("'kernelGenerateRandoms' took {:f} [ms]", timer.getMilliseconds());
    return timer.getMilliseconds();
}

float pseudo__random_subsampling::initRandoms(unsigned int seed, unique_ptr<CudaArray<hiprandState_t>> &states, uint32_t nodeAmount) {

    // Calculate kernel dimensions
    dim3 grid, block;
    tools::create1DKernel(block, grid, nodeAmount);

    // Initial point counting
    tools::KernelTimer timer;
    timer.start();
    kernelInitRandoms << < grid, block >> > (seed, states->devicePointer(), nodeAmount);
    timer.stop();
    gpuErrchk(hipGetLastError());

    spdlog::debug("'kernelInitRandoms' took {:f} [ms]", timer.getMilliseconds());
    return timer.getMilliseconds();
}

float pseudo__random_subsampling::subsample(
        unique_ptr<CudaArray<uint8_t>> &cloud,
        unique_ptr<CudaArray<uint32_t>> &cloudDataLUT,
        uint32_t dataLUTStartIndex,
        unique_ptr<CudaArray<uint32_t>> &countingGrid,
        unique_ptr<CudaArray<int>> &denseToSparseLUT,
        unique_ptr<CudaArray<uint32_t>> &sparseIndexCounter,
        PointCloudMetadata metadata,
        uint32_t gridSideLength
) {
    // Calculate kernel dimensions
    dim3 grid, block;
    tools::create1DKernel(block, grid, metadata.pointAmount);

    // Initial point counting
    tools::KernelTimer timer;
    timer.start();
    kernelSubsample << < grid, block >> > (
            cloud->devicePointer(),
                    cloudDataLUT->devicePointer(),
                    dataLUTStartIndex,
                    countingGrid->devicePointer(),
                    denseToSparseLUT->devicePointer(),
                    sparseIndexCounter->devicePointer(),
                    metadata,
                    gridSideLength);
    timer.stop();
    gpuErrchk(hipGetLastError());

    spdlog::debug("'kernelSubsample' took {:f} [ms]", timer.getMilliseconds());
    return timer.getMilliseconds();
}