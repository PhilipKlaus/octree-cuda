#include <sparseOctree.h>

#include <subsample_evaluating.cuh>
#include <random_subsampling.cuh>

std::tuple<float, float> SparseOctree::randomSubsampling(
        const unique_ptr<Chunk[]> &h_octreeSparse,
        const unique_ptr<int[]> &h_sparseToDenseLUT,
        uint32_t sparseVoxelIndex,
        uint32_t level,
        unique_ptr<CudaArray<uint32_t>> &subsampleCountingGrid,
        unique_ptr<CudaArray<int>> &subsampleDenseToSparseLUT,
        unique_ptr<CudaArray<uint32_t>> &subsampleSparseVoxelCount,
        unique_ptr<CudaArray<hiprandState_t >> &randomStates,
        unique_ptr<CudaArray<uint32_t >> &randomIndices) {

    Chunk voxel = h_octreeSparse[sparseVoxelIndex];
    std::tuple<float, float> accumulatedTime = {0,0};

    // 1. Depth first traversal
    for(int childIndex : voxel.childrenChunks) {
        if(childIndex != -1) {
            std::tuple<float, float> childTime = randomSubsampling(
                    h_octreeSparse,
                    h_sparseToDenseLUT,
                    childIndex,
                    level - 1,
                    subsampleCountingGrid,
                    subsampleDenseToSparseLUT,
                    subsampleSparseVoxelCount,
                    randomStates,
                    randomIndices);

            get<0>(accumulatedTime) += get<0>(childTime);
            get<1>(accumulatedTime) += get<1>(childTime);
        }
    }

    // 2. Now we can assure that all direct children have subsamples
    if(voxel.isParent) {

        // 3. Calculate the dense coordinates of the voxel
        BoundingBox bb{};
        CoordinateVector<uint32_t> coords{};
        auto denseVoxelIndex = h_sparseToDenseLUT[sparseVoxelIndex];
        calculateVoxelBB(bb, coords, denseVoxelIndex, level);

        PointCloudMetadata metadata = itsMetadata.cloudMetadata;
        metadata.boundingBox = bb;
        metadata.cloudOffset = bb.minimum;

        // 4. Pre-calculate the subsamples and count the subsampled points
        for(int childIndex : voxel.childrenChunks) {

            if(childIndex != -1) {
                Chunk child = h_octreeSparse[childIndex];
                metadata.pointAmount = child.isParent ? itsSubsampleLUTs[childIndex]->pointCount() : child.pointCount;

                get<0>(accumulatedTime) += subsampling::evaluateSubsamples<float>(
                        itsCloudData,
                        child.isParent ? itsSubsampleLUTs[childIndex] : itsDataLUT,
                        child.isParent ? 0 : child.chunkDataIndex,
                        subsampleCountingGrid,
                        subsampleDenseToSparseLUT,
                        subsampleSparseVoxelCount,
                        metadata,
                        itsMetadata.subsamplingGrid);
            }
        }

        // 5. Reserve memory for a data LUT for the parent node
        auto amountUsedVoxels = subsampleSparseVoxelCount->toHost()[0];

        //---------- GENERATE RANDOM INDICES FOR SUBSAMPLING ----------------
        get<1>(accumulatedTime) += subsampling::generateRandoms(randomStates, randomIndices, subsampleDenseToSparseLUT, subsampleCountingGrid, subsampleDenseToSparseLUT->pointCount());


        auto subsampleLUT = make_unique<CudaArray<uint32_t >>(amountUsedVoxels, "subsampleLUT_" + to_string(sparseVoxelIndex));
        itsSubsampleLUTs.insert(make_pair(sparseVoxelIndex, move(subsampleLUT)));

        // 6. Distribute points to the parent data LUT
        for(int childIndex : voxel.childrenChunks) {

            if(childIndex != -1) {
                Chunk child = h_octreeSparse[childIndex];
                metadata.pointAmount = child.isParent ? itsSubsampleLUTs[childIndex]->pointCount() : child.pointCount;

                get<1>(accumulatedTime) += subsampling::randomPointSubsample<float>(
                        itsCloudData,
                        child.isParent ? itsSubsampleLUTs[childIndex] : itsDataLUT,
                        child.isParent ? 0 : child.chunkDataIndex,
                        itsSubsampleLUTs[sparseVoxelIndex],
                        subsampleCountingGrid,
                        subsampleDenseToSparseLUT,
                        subsampleSparseVoxelCount,
                        metadata,
                        itsMetadata.subsamplingGrid,
                        randomIndices);
            }
        }
    }
    return accumulatedTime;
}