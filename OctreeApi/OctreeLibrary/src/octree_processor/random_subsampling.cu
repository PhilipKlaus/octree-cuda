#include "hip/hip_runtime.h"
#include "kernel_executor.cuh"
#include "octree_processor.h"
#include "random_subsampling.cuh"
#include "subsample_evaluating.cuh"


float OctreeProcessor::initRandomStates (unsigned int seed, GpuRandomState& states, uint32_t nodeAmount)
{
    return executeKernel (subsampling::kernelInitRandoms, nodeAmount, seed, states->devicePointer (), nodeAmount);
}


SubsamplingTimings OctreeProcessor::randomSubsampling (
        const unique_ptr<Chunk[]>& h_octreeSparse,
        const unique_ptr<int[]>& h_sparseToDenseLUT,
        uint32_t sparseVoxelIndex,
        uint32_t level,
        GpuArrayU32& subsampleCountingGrid,
        GpuAveraging & averagingGrid,
        GpuArrayI32& subsampleDenseToSparseLUT,
        GpuArrayU32& subsampleSparseVoxelCount,
        GpuRandomState& randomStates,
        GpuArrayU32& randomIndices)
{
    PointCloudMetadata cloudMetadata = itsMetadata.cloudMetadata;

    Chunk voxel                = h_octreeSparse[sparseVoxelIndex];
    SubsamplingTimings timings = {};

    // Depth first traversal
    for (int childIndex : voxel.childrenChunks)
    {
        if (childIndex != -1)
        {
            SubsamplingTimings childTiming = randomSubsampling (
                    h_octreeSparse,
                    h_sparseToDenseLUT,
                    childIndex,
                    level - 1,
                    subsampleCountingGrid,
                    averagingGrid,
                    subsampleDenseToSparseLUT,
                    subsampleSparseVoxelCount,
                    randomStates,
                    randomIndices);

            timings.subsampleEvaluation += childTiming.subsampleEvaluation;
            timings.generateRandoms += childTiming.generateRandoms;
            timings.averaging += childTiming.averaging;
            timings.subsampling += childTiming.subsampling;
        }
    }

    // Now we can assure that all direct children have subsamples
    if (voxel.isParent)
    {
        // Prepare and update the SubsampleConfig on the GPU
        SubsampleSet subsampleSet{};
        uint32_t maxPoints = prepareSubsampleConfig (subsampleSet, voxel, h_octreeSparse);

        // Parent bounding box calculation
        PointCloudMetadata metadata = cloudMetadata;
        auto denseVoxelIndex        = h_sparseToDenseLUT[sparseVoxelIndex];
        calculateVoxelBB (metadata, denseVoxelIndex, level);

        Kernel::KernelConfig kernelConfig      = {metadata.cloudType, maxPoints};
        KernelStructs::Cloud cloud       = {itsCloud->getCloudDevice (), 0, metadata.pointDataStride};
        KernelStructs::Gridding gridding = {itsSubsampleMetadata.subsamplingGrid, metadata.cubicSize (), metadata.bbCubic.min};

        // Evaluate how many points fall in each cell
        timings.subsampleEvaluation += Kernel::evaluateSubsamples (
                kernelConfig,
                subsampleSet,
                subsampleCountingGrid->devicePointer (),
                averagingGrid->devicePointer(),
                subsampleDenseToSparseLUT->devicePointer (),
                subsampleSparseVoxelCount->devicePointer (),
                cloud,
                gridding);

        // Prepare one random point index per cell
        uint32_t threads = subsampleDenseToSparseLUT->pointCount ();
        timings.generateRandoms += executeKernel (
                subsampling::kernelGenerateRandoms,
                threads,
                randomStates->devicePointer (),
                randomIndices->devicePointer (),
                subsampleDenseToSparseLUT->devicePointer (),
                subsampleSparseVoxelCount->devicePointer (),
                subsampleCountingGrid->devicePointer (),
                threads);

        // Reserve memory for a data LUT for the parent node
        auto amountUsedVoxels = subsampleSparseVoxelCount->toHost ()[0];
        // Create LUT and averaging data for parent node
        auto subsampleLUT  = createGpuU32 (amountUsedVoxels, "subsampleLUT_" + to_string (sparseVoxelIndex));
        auto averagingData = createGpuAveraging (amountUsedVoxels, "averagingData_" + to_string (sparseVoxelIndex));
        averagingData->memset(0);
        itsParentLut.insert (make_pair (sparseVoxelIndex, move (subsampleLUT)));
        itsAveragingData.insert (make_pair (sparseVoxelIndex, move (averagingData)));

        // Distribute the subsampled points in parallel for all child nodes
        timings.subsampling += Kernel::randomPointSubsampling (
                kernelConfig,
                subsampleSet,
                itsParentLut[sparseVoxelIndex]->devicePointer (),
                itsAveragingData[sparseVoxelIndex]->devicePointer (),
                subsampleCountingGrid->devicePointer (),
                averagingGrid->devicePointer(),
                subsampleDenseToSparseLUT->devicePointer (),
                subsampleSparseVoxelCount->devicePointer (),
                cloud,
                gridding,
                randomIndices->devicePointer (),
                itsSubsampleMetadata.useReplacementScheme);
    }

    return timings;
}
