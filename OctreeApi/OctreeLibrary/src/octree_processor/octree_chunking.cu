#include "hip/hip_runtime.h"
/**
 * @file octree_chunking.cu
 * @author Philip Klaus
 * @brief Contains implementations of chunking-related Octreeprocessor methods
 */

#include "kernel_executor.cuh"
#include "octree_processpr_impl.cuh"

#include "hierarchical_merging.cuh"
#include "octree_initialization.cuh"
#include "point_count_propagation.cuh"
#include "point_counting.cuh"
#include "point_distributing.cuh"


void OctreeProcessorPimpl::OctreeProcessorImpl::initialPointCounting ()
{
    // Allocate the dense point count
    itsDensePointCountPerVoxel = createGpuU32 (itsMetadata.nodeAmountDense, "DensePointCountPerVoxel");
    itsDensePointCountPerVoxel->memset (0);

    // Allocate the conversion LUT from dense to sparse
    itsDenseToSparseLUT = createGpuI32 (itsMetadata.nodeAmountDense, "DenseToSparseLUT");
    itsDenseToSparseLUT->memset (-1);

    // Allocate the temporary sparseIndexCounter
    itsTmpCounting = createGpuU32 (1, "nodeAmountSparse");
    itsTmpCounting->memset (0);

    auto& meta                       = itsCloud->getMetadata ();
    Kernel::KernelConfig config      = {meta.cloudType, meta.pointAmount};
    KernelStructs::Cloud cloud       = {itsCloud->getCloudDevice (), meta.pointAmount, meta.pointDataStride};
    KernelStructs::Gridding gridding = {itsOctreeData->getGridSize (0), meta.cubicSize (), meta.bbCubic.min};

    float time = Kernel::pointCounting (
            config,
            itsDensePointCountPerVoxel->devicePointer (),
            itsTmpCounting->devicePointer (),
            itsDenseToSparseLUT->devicePointer (),
            cloud,
            gridding);

    itsTimeMeasurement.emplace_back ("kernelPointCounting", time);
    spdlog::info ("[kernel] kernelPointCounting took {:f} [ms]", time);
}

void OctreeProcessorPimpl::OctreeProcessorImpl::performCellMerging ()
{
    float timeAccumulated = 0;

    // Perform a hierarchicaly merging of the grid cells which results in an octree structure
    for (uint32_t i = 0; i < itsMetadata.depth; ++i)
    {
        float time = executeKernel (
                chunking::kernelPropagatePointCounts,
                itsOctreeData->getNodes (i + 1),
                itsDensePointCountPerVoxel->devicePointer (),
                itsDenseToSparseLUT->devicePointer (),
                itsTmpCounting->devicePointer (),
                itsOctreeData->getNodes (i + 1),
                itsOctreeData->getGridSize (i + 1),
                itsOctreeData->getGridSize (i),
                itsOctreeData->getNodeOffset (i + 1),
                itsOctreeData->getNodeOffset (i));

        itsTimeMeasurement.emplace_back (
                "kernelPropagatePointCounts_" + std::to_string (itsOctreeData->getGridSize (i)), time);
        timeAccumulated += time;
    }

    spdlog::info ("[kernel] kernelPropagatePointCounts took {:f}[ms]", timeAccumulated);

    // Retrieve the actual amount of sparse nodes in the octree and allocate the octree data structure
    itsMetadata.nodeAmountSparse = itsTmpCounting->toHost ()[0];
    itsOctreeData->createOctree (itsMetadata.nodeAmountSparse);
    // Allocate the conversion LUT from sparse to dense
    itsSparseToDenseLUT = createGpuI32 (itsMetadata.nodeAmountSparse, "sparseToDenseLUT");
    itsSparseToDenseLUT->memset (-1);

    initLowestOctreeHierarchy ();
    mergeHierarchical ();
}

void OctreeProcessorPimpl::OctreeProcessorImpl::initLowestOctreeHierarchy ()
{
    float time = executeKernel (
            chunking::kernelInitLeafNodes,
            itsOctreeData->getNodes (0),
            itsOctreeData->getDevice (),
            itsDensePointCountPerVoxel->devicePointer (),
            itsDenseToSparseLUT->devicePointer (),
            itsSparseToDenseLUT->devicePointer (),
            itsOctreeData->getNodes (0));

    itsTimeMeasurement.emplace_back ("kernelInitLeafNodes", time);
    spdlog::info ("[kernel] kernelInitLeafNodes took {:f}[ms]", time);
}


void OctreeProcessorPimpl::OctreeProcessorImpl::mergeHierarchical ()
{
    itsTmpCounting->memset (0);

    float timeAccumulated = 0;
    for (uint32_t i = 0; i < itsMetadata.depth; ++i)
    {
        float time = executeKernel (
                chunking::kernelMergeHierarchical,
                itsOctreeData->getNodes (i + 1),
                itsOctreeData->getDevice (),
                itsDensePointCountPerVoxel->devicePointer (),
                itsDenseToSparseLUT->devicePointer (),
                itsSparseToDenseLUT->devicePointer (),
                itsTmpCounting->devicePointer (),
                itsMetadata.mergingThreshold,
                itsOctreeData->getNodes (i + 1),
                itsOctreeData->getGridSize (i + 1),
                itsOctreeData->getGridSize (i),
                itsOctreeData->getNodeOffset (i + 1),
                itsOctreeData->getNodeOffset (i));

        timeAccumulated += time;
        itsTimeMeasurement.emplace_back (
                "kernelMergeHierarchical_" + std::to_string (itsOctreeData->getGridSize (i)), time);
    }

    spdlog::info ("[kernel] kernelMergeHierarchical took {:f}[ms]", timeAccumulated);
}

void OctreeProcessorPimpl::OctreeProcessorImpl::distributePoints ()
{
    auto tmpIndexRegister = createGpuU32 (itsMetadata.nodeAmountSparse, "tmpIndexRegister");
    tmpIndexRegister->memset (0);

    auto& meta                       = itsCloud->getMetadata ();
    Kernel::KernelConfig config      = {meta.cloudType, meta.pointAmount};
    KernelStructs::Cloud cloud       = {itsCloud->getCloudDevice (), meta.pointAmount, meta.pointDataStride};
    KernelStructs::Gridding gridding = {itsOctreeData->getGridSize (0), meta.cubicSize (), meta.bbCubic.min};

    float time = Kernel::distributePoints (
            config,
            itsOctreeData->getDevice (),
            itsLeafLut->devicePointer (),
            itsDenseToSparseLUT->devicePointer (),
            tmpIndexRegister->devicePointer (),
            cloud,
            gridding);

    itsTimeMeasurement.emplace_back ("kernelDistributePoints", time);
    spdlog::info ("[kernel] kernelDistributePoints took {:f}[ms]", time);
}
