#include "hip/hip_runtime.h"
//
// Created by KlausP on 04.11.2020.
//

#include "octree_processor.h"
#include "ply_exporter.cuh"
#include "potree_exporter.cuh"
#include "tools.cuh"

OctreeProcessor::OctreeProcessor (
        uint8_t* pointCloud,
        uint32_t chunkingGrid,
        uint32_t mergingThreshold,
        PointCloudMetadata cloudMetadata,
        SubsampleMetadata subsamplingMetadata)
{
    itsOctreeData = std::make_unique<OctreeData> (chunkingGrid);

    // Initialize metadata
    itsMetadata                  = {};
    itsMetadata.depth            = itsOctreeData->getDepth ();
    itsMetadata.nodeAmountDense  = itsOctreeData->getOverallNodes ();
    itsMetadata.chunkingGrid     = chunkingGrid;
    itsMetadata.mergingThreshold = mergingThreshold;
    itsMetadata.cloudMetadata    = cloudMetadata;
    itsSubsampleMetadata         = subsamplingMetadata;

    if (cloudMetadata.memoryType == CLOUD_HOST)
    {
        itsCloud = std::make_unique<PointCloudHost> (pointCloud, cloudMetadata);
    }
    else
    {
        itsCloud = std::make_unique<PointCloudDevice> (pointCloud, cloudMetadata);
    }

    // Create data LUT
    itsLeafLut = createGpuU32 (cloudMetadata.pointAmount, "Data LUT");
    spdlog::info ("Prepared empty SparseOctree");
}

void OctreeProcessor::calculateVoxelBB (PointCloudMetadata& metadata, uint32_t denseVoxelIndex, uint32_t level)
{
    Vector3<uint32_t> coords = {};

    // 1. Calculate coordinates of voxel within the actual level
    auto indexInLevel = denseVoxelIndex - itsOctreeData->getNodeOffset (level);
    tools::mapFromDenseIdxToDenseCoordinates (coords, indexInLevel, itsOctreeData->getGridSize (level));

    // 2. Calculate the bounding box for the actual voxel
    // ToDo: Include scale and offset!!!
    double min      = itsMetadata.cloudMetadata.bbCubic.min.x;
    double max      = itsMetadata.cloudMetadata.bbCubic.max.x;
    double side     = max - min;
    auto cubicWidth = side / itsOctreeData->getGridSize (level);

    metadata.bbCubic.min.x = itsMetadata.cloudMetadata.bbCubic.min.x + coords.x * cubicWidth;
    metadata.bbCubic.min.y = itsMetadata.cloudMetadata.bbCubic.min.y + coords.y * cubicWidth;
    metadata.bbCubic.min.z = itsMetadata.cloudMetadata.bbCubic.min.z + coords.z * cubicWidth;
    metadata.bbCubic.max.x = metadata.bbCubic.min.x + cubicWidth;
    metadata.bbCubic.max.y = metadata.bbCubic.min.y + cubicWidth;
    metadata.bbCubic.max.z = metadata.bbCubic.min.z + cubicWidth;
    metadata.cloudOffset   = metadata.bbCubic.min;
}

// ToDo: call appropriate export function!!!
void OctreeProcessor::exportPlyNodes (const string& folderPath)
{
    auto start = std::chrono::high_resolution_clock::now ();
    /*PlyExporter<coordinateType, colorType> plyExporter (
            itsCloudData, itsOctree, itsDataLUT, itsSubsampleLUTs, itsAveragingData, itsMetadata);
    plyExporter.exportOctree (folderPath);*/
    PotreeExporter<double, uint8_t> potreeExporter (
            itsCloud,
            itsOctreeData->getHost (),
            itsLeafLut,
            itsParentLut,
            itsAveragingData,
            itsMetadata,
            itsSubsampleMetadata);
    auto finish                           = std::chrono::high_resolution_clock::now ();
    std::chrono::duration<double> elapsed = finish - start;
    spdlog::info ("Copy from device to host tooks {} seconds", elapsed.count ());

    start = std::chrono::high_resolution_clock::now ();
    potreeExporter.exportOctree (folderPath);
    finish  = std::chrono::high_resolution_clock::now ();
    elapsed = finish - start;
    spdlog::info ("Export tooks {} seconds", elapsed.count ());
    itsTimeMeasurement.emplace_back ("exportPotree", elapsed.count () * 1000);
}
