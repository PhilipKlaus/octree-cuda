#include "hip/hip_runtime.h"
#include "kernel_executor.cuh"
#include "octree_processor.h"
#include "random_subsampling.cuh"
#include "subsample_evaluating.cuh"


void OctreeProcessor::performSubsampling ()
{
    auto h_octreeSparse     = itsOctreeData->getHost ();
    auto h_sparseToDenseLUT = itsSparseToDenseLUT->toHost ();
    auto nodesBaseLevel     = static_cast<uint32_t> (pow (itsSubsampleMetadata.subsamplingGrid, 3.f));

    // Prepare data strucutres for the subsampling
    auto pointCountGrid  = createGpuU32 (nodesBaseLevel, "pointCountGrid");
    auto averagingGrid   = createGpuAveraging (nodesBaseLevel, "averagingGrid");
    auto denseToSpareLUT = createGpuI32 (nodesBaseLevel, "denseToSpareLUT");
    auto voxelCount      = createGpuU32 (1, "voxelCount");

    pointCountGrid->memset (0);
    denseToSpareLUT->memset (-1);
    voxelCount->memset (0);

    SubsamplingTimings timings = {};

    auto randomStates = createGpuRandom (1024, "randomStates");

    // ToDo: Time measurement
    executeKernel (subsampling::kernelInitRandoms, 1024, std::time (0), randomStates->devicePointer (), 1024);
    auto randomIndices = createGpuU32 (nodesBaseLevel, "randomIndices");

    timings = randomSubsampling (
            h_octreeSparse,
            h_sparseToDenseLUT,
            getRootIndex (),
            itsMetadata.depth,
            pointCountGrid,
            averagingGrid,
            denseToSpareLUT,
            voxelCount,
            randomStates,
            randomIndices);


    itsTimeMeasurement.emplace_back ("subsampleEvaluation", timings.subsampleEvaluation);
    itsTimeMeasurement.emplace_back ("generateRandoms", timings.generateRandoms);
    itsTimeMeasurement.emplace_back ("subsampling", timings.subsampling);
    spdlog::info ("[kernel] kernelEvaluateSubsamples took: {}[ms]", timings.subsampleEvaluation);
    spdlog::info ("[kernel] kernelGenerateRandoms took: {}[ms]", timings.generateRandoms);
    spdlog::info ("[kernel] kernelRandomPointSubsample took: {}[ms]", timings.subsampling);
}


SubsamplingTimings OctreeProcessor::randomSubsampling (
        const shared_ptr<Chunk[]>& h_octreeSparse,
        const unique_ptr<int[]>& h_sparseToDenseLUT,
        uint32_t sparseVoxelIndex,
        uint32_t level,
        GpuArrayU32& subsampleCountingGrid,
        GpuAveraging& averagingGrid,
        GpuArrayI32& subsampleDenseToSparseLUT,
        GpuArrayU32& subsampleSparseVoxelCount,
        GpuRandomState& randomStates,
        GpuArrayU32& randomIndices)
{
    PointCloudMetadata cloudMetadata = itsMetadata.cloudMetadata;

    Chunk voxel                = h_octreeSparse[sparseVoxelIndex];
    SubsamplingTimings timings = {};

    // Depth first traversal
    for (int childIndex : voxel.childrenChunks)
    {
        if (childIndex != -1)
        {
            SubsamplingTimings childTiming = randomSubsampling (
                    h_octreeSparse,
                    h_sparseToDenseLUT,
                    childIndex,
                    level - 1,
                    subsampleCountingGrid,
                    averagingGrid,
                    subsampleDenseToSparseLUT,
                    subsampleSparseVoxelCount,
                    randomStates,
                    randomIndices);

            timings.subsampleEvaluation += childTiming.subsampleEvaluation;
            timings.generateRandoms += childTiming.generateRandoms;
            timings.subsampling += childTiming.subsampling;
        }
    }

    // Now we can assure that all direct children have subsamples
    if (voxel.isParent)
    {
        // Prepare and update the SubsampleConfig on the GPU
        SubsampleSet subsampleSet{};
        uint32_t maxPoints = prepareSubsampleConfig (subsampleSet, voxel, h_octreeSparse);

        // Parent bounding box calculation
        PointCloudMetadata metadata = cloudMetadata;
        auto denseVoxelIndex        = h_sparseToDenseLUT[sparseVoxelIndex];
        calculateVoxelBB (metadata, denseVoxelIndex, level);

        Kernel::KernelConfig kernelConfig = {metadata.cloudType, maxPoints};
        KernelStructs::Cloud cloud        = {itsCloud->getCloudDevice (), 0, metadata.pointDataStride};
        KernelStructs::Gridding gridding  = {
                itsSubsampleMetadata.subsamplingGrid, metadata.cubicSize (), metadata.bbCubic.min};

        // Evaluate how many points fall in each cell
        timings.subsampleEvaluation += Kernel::evaluateSubsamples (
                kernelConfig,
                subsampleSet,
                subsampleCountingGrid->devicePointer (),
                averagingGrid->devicePointer (),
                subsampleDenseToSparseLUT->devicePointer (),
                subsampleSparseVoxelCount->devicePointer (),
                cloud,
                gridding);

        // Prepare one random point index per cell
        uint32_t threads = subsampleDenseToSparseLUT->pointCount ();
        timings.generateRandoms += executeKernel (
                subsampling::kernelGenerateRandoms,
                threads,
                randomStates->devicePointer (),
                randomIndices->devicePointer (),
                subsampleDenseToSparseLUT->devicePointer (),
                subsampleSparseVoxelCount->devicePointer (),
                subsampleCountingGrid->devicePointer (),
                threads);

        // Reserve memory for a data LUT for the parent node
        auto amountUsedVoxels = subsampleSparseVoxelCount->toHost ()[0];
        // Create LUT and averaging data for parent node
        auto subsampleLUT  = createGpuU32 (amountUsedVoxels, "subsampleLUT_" + to_string (sparseVoxelIndex));
        auto averagingData = createGpuAveraging (amountUsedVoxels, "averagingData_" + to_string (sparseVoxelIndex));
        averagingData->memset (0);
        itsParentLut.insert (make_pair (sparseVoxelIndex, move (subsampleLUT)));
        itsAveragingData.insert (make_pair (sparseVoxelIndex, move (averagingData)));

        // Distribute the subsampled points in parallel for all child nodes
        timings.subsampling += Kernel::randomPointSubsampling (
                kernelConfig,
                subsampleSet,
                itsParentLut[sparseVoxelIndex]->devicePointer (),
                itsAveragingData[sparseVoxelIndex]->devicePointer (),
                subsampleCountingGrid->devicePointer (),
                averagingGrid->devicePointer (),
                subsampleDenseToSparseLUT->devicePointer (),
                subsampleSparseVoxelCount->devicePointer (),
                cloud,
                gridding,
                randomIndices->devicePointer (),
                itsSubsampleMetadata.useReplacementScheme);
    }

    return timings;
}


uint32_t OctreeProcessor::prepareSubsampleConfig (
        SubsampleSet& subsampleSet, Chunk& voxel, const shared_ptr<Chunk[]>& h_octreeSparse)
{
    uint32_t maxPoints = 0;
    auto* config       = (SubsampleConfig*)(&subsampleSet);

    for (uint8_t i = 0; i < 8; ++i)
    {
        int childIndex = voxel.childrenChunks[i];
        if (childIndex != -1)
        {
            Chunk child               = h_octreeSparse[childIndex];
            config[i].pointAmount     = child.isParent ? itsParentLut[childIndex]->pointCount () : child.pointCount;
            maxPoints                 = max (maxPoints, config[i].pointAmount);
            config[i].averagingAdress = child.isParent ? itsAveragingData[childIndex]->devicePointer () : nullptr;
            config[i].lutStartIndex   = child.isParent ? 0 : child.chunkDataIndex;
            config[i].lutAdress =
                    child.isParent ? itsParentLut[childIndex]->devicePointer () : itsLeafLut->devicePointer ();
        }
        else
        {
            config[i].pointAmount     = 0;
            config[i].averagingAdress = nullptr;
            config[i].lutAdress       = nullptr;
        }
    }
    return maxPoints;
}
